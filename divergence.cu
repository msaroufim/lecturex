
#include <hip/hip_runtime.h>
__global__ void processArrayWithDivergence(int *data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (data[idx] % 2 == 0) {
            data[idx] = data[idx] * 2; // Operation for even numbers
        } else {
            data[idx] = data[idx] + 1; // Operation for odd numbers
        }
    }
}

__global__ void processArrayWithoutDivergence(int *data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int isEven = !(data[idx] % 2); // 1 for even, 0 for odd
        data[idx] = isEven * (data[idx] * 2) + (!isEven) * (data[idx] + 1);
    }
}
