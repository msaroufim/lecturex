
#include <hip/hip_runtime.h>
__global__ void copyDataNonCoalesced(float *in, float *out, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Access pattern might not be coalesced depending on the index calculation
        out[index] = in[(index * 2) % n]; // Example of a non-coalesced access
    }
}


__global__ void copyDataCoalesced(float *in, float *out, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Ensure coalesced memory access by accessing consecutive memory addresses
        out[index] = in[index]; // Direct 1-to-1 mapping ensures coalesced access
    }
}
